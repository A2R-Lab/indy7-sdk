#include <filesystem>
#include <chrono>
#include <fstream>
#include <thread>
#include <mutex>
#include <atomic>
#include <algorithm>
#include "rclcpp/rclcpp.hpp"
#include "indy7_msgs/msg/joint_state.hpp"
#include "indy7_msgs/msg/joint_trajectory.hpp"
#include "indy7_msgs/msg/joint_trajectory_point.hpp"
#include "trajopt_solver.cuh"
#include "csv_utils.h"

namespace msgs = indy7_msgs::msg;
using Clock = std::chrono::system_clock;
using TimePoint = std::chrono::time_point<Clock>;
using Duration = std::chrono::duration<double>;
using SimTime = rclcpp::Time;

class TrajoptNode : public rclcpp::Node
{
public:
    explicit TrajoptNode(const std::string& traj_file)
        : Node("trajopt_node")
        , timestep_(Duration(0.01))
        , pcg_exit_tol_(5e-4)
        , pcg_max_iter_(173)
        , state_updated_(false)
        , warm_start_complete_(false)
        , optimization_in_progress_(false)
        , use_sim_time_(false)
    {
        use_sim_time_ = get_parameter("use_sim_time").as_bool();
        RCLCPP_INFO(this->get_logger(), "Using %s time", use_sim_time_ ? "simulation" : "system");
        RCLCPP_INFO(this->get_logger(), "Initializing TrajoptNode");

        setupCommunication();
        initializeSolver(traj_file);
        waitForInitialState();
    }

    ~TrajoptNode() {
        RCLCPP_INFO(this->get_logger(), "Shutting down TrajoptNode");
    }

private:
    void setupCommunication() {
        state_sub_ = create_subscription<msgs::JointState>(
            "joint_states", 1, 
            std::bind(&TrajoptNode::stateCallback, this, std::placeholders::_1)
        );
        traj_pub_ = create_publisher<msgs::JointTrajectory>("joint_trajectory", 1);
    }

    void initializeSolver(const std::string& traj_file) {
        std::vector<float> goal_eePos_traj_1d = readCsvToVector<float>(traj_file);
        solver_ = std::make_unique<TrajoptSolver<float>>(
            goal_eePos_traj_1d,
            timestep_.count(),
            pcg_exit_tol_,
            pcg_max_iter_
        );
        RCLCPP_INFO(this->get_logger(), "Solver initialized");

        // Pre-allocate trajectory message
        traj_msg_.knot_points = solver_->numKnotPoints();
        traj_msg_.points.resize(solver_->numKnotPoints());
        full_state_.reserve(solver_->stateSize());
    }

    void waitForInitialState() {
        RCLCPP_INFO(this->get_logger(), "Waiting for initial state...");
        while (rclcpp::ok() && !state_updated_) {
            rclcpp::spin_some(this->get_node_base_interface());
            std::this_thread::sleep_for(std::chrono::milliseconds(10));
        }

        std::vector<float> current_joint_positions(
            current_state_.positions.begin(), 
            current_state_.positions.end()
        );
        solver_->initializeXU(current_joint_positions);
        
        RCLCPP_INFO(this->get_logger(), "Received initial state, starting solver warm start...");
        solver_->warmStart();
        hipDeviceSynchronize();
        warm_start_complete_ = true;
        RCLCPP_INFO(this->get_logger(), "Warm start complete");
        initializeStartTime();
    }

    void initializeStartTime() {
        if (use_sim_time_) {
            while (rclcpp::ok() && current_state_.header.stamp.sec == 0) {
                rclcpp::spin_some(this->get_node_base_interface());
                std::this_thread::sleep_for(std::chrono::milliseconds(10));
            }
            trajectory_start_stamp_ = current_state_.header.stamp;
        } else {
            trajectory_start_time_ = Clock::now();
        }
    }

    void stateCallback(const std::shared_ptr<const msgs::JointState>& msg) {
        if (optimization_in_progress_.load() || solver_->isTrajectoryComplete()) {
            return;
        }

        updateCurrentState(msg);
        if (!warm_start_complete_) { return; }
        optimization_in_progress_ = true;
        runOptimization(msg);
        optimization_in_progress_ = false;
    }

    void updateCurrentState(const std::shared_ptr<const msgs::JointState>& msg) {
        std::lock_guard<std::mutex> lock(state_mutex_);
        current_state_ = *msg;
        state_updated_ = true;

        full_state_.clear();
        full_state_.insert(full_state_.end(), 
            current_state_.positions.begin(), 
            current_state_.positions.end());
        full_state_.insert(full_state_.end(), 
            current_state_.velocities.begin(), 
            current_state_.velocities.end());
    }

    void runOptimization(const std::shared_ptr<const msgs::JointState>& msg) {
        solver_->shiftTrajectory(full_state_, calculateElapsedTime(msg->header.stamp));
        
        std::string stats = solver_->runTrajoptIteration();
        RCLCPP_INFO(this->get_logger(), "Optimization stats: %s", stats.c_str());
        RCLCPP_INFO(this->get_logger(), "Trajectory offset: %u", solver_->getTrajectoryOffset());

        publishTrajectory(msg->header.stamp);
    }

    double calculateElapsedTime(const builtin_interfaces::msg::Time& current_stamp) const {
        if (use_sim_time_) {
            SimTime current_time(current_stamp);
            SimTime start_time(trajectory_start_stamp_);
            return (current_time - start_time).seconds();
        }
        return Duration(Clock::now() - trajectory_start_time_).count();
    }

    void publishTrajectory(const builtin_interfaces::msg::Time& stamp) {
        auto traj_msg = msgs::JointTrajectory();
        traj_msg.header.stamp = stamp;
        traj_msg.knot_points = solver_->numKnotPoints();
        traj_msg.dt = timestep_.count();

        const auto [traj_data, traj_size] = solver_->getOptimizedTrajectory();
        const int stride = solver_->stateSize() + solver_->controlSize();
        
        traj_msg.points.clear();
        for (int i = 0; i < traj_msg.knot_points; i++) {
            msgs::JointTrajectoryPoint point;
            for (size_t j = 0; j < 6; ++j) {
                point.positions[j] = traj_data[i * stride + j];
                point.velocities[j] = traj_data[i * stride + solver_->stateSize()/2 + j];
                point.torques[j] = traj_data[i * stride + solver_->stateSize() + j];
            }
            traj_msg.points.emplace_back(point);
        }

        traj_pub_->publish(traj_msg);
    }

    // ROS communication
    rclcpp::Subscription<msgs::JointState>::SharedPtr state_sub_;
    rclcpp::Publisher<msgs::JointTrajectory>::SharedPtr traj_pub_;

    // Solver configuration
    std::unique_ptr<TrajoptSolver<float, 12, 6, 128, 128>> solver_;
    const Duration timestep_;
    const float pcg_exit_tol_;
    const int pcg_max_iter_;

    // State management
    msgs::JointState current_state_;
    std::vector<float> full_state_;
    std::atomic<bool> state_updated_;
    std::mutex state_mutex_;

    // Trajectory management
    msgs::JointTrajectory traj_msg_;
    TimePoint trajectory_start_time_;
    builtin_interfaces::msg::Time trajectory_start_stamp_;

    // Control flags
    std::atomic<bool> warm_start_complete_;
    std::atomic<bool> optimization_in_progress_;
    bool use_sim_time_;
};

int main(int argc, char* argv[])
{
    rclcpp::init(argc, argv);
    auto node = std::make_shared<TrajoptNode>(argv[1]);
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}